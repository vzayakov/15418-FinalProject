#include "hip/hip_runtime.h"
// Main CUDA file for our parallel terrain generation algorithm
// Authors: Petros Emmanouilidis and Victor Zayakov

#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>
#include <cstdlib>

#include "noiseMap.h"
#include "util.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

// Permutation table for Perlin noise
short permutationGlobal[256] = { 151,160,137,91,90,15,
   131,13,201,95,96,53,194,233,7,225,140,36,103,30,69,142,8,99,37,240,21,10,23,
   190, 6,148,247,120,234,75,0,26,197,62,94,252,219,203,117,35,11,32,57,177,33,
   88,237,149,56,87,174,20,125,136,171,168, 68,175,74,165,71,134,139,48,27,166,
   77,146,158,231,83,111,229,122,60,211,133,230,220,105,92,41,55,46,245,40,244,
   102,143,54, 65,25,63,161, 1,216,80,73,209,76,132,187,208, 89,18,169,200,196,
   135,130,116,188,159,86,164,100,109,198,173,186, 3,64,52,217,226,250,124,123,
   5,202,38,147,118,126,255,82,85,212,207,206,59,227,47,16,58,17,182,189,28,42,
   223,183,170,213,119,248,152, 2,44,154,163, 70,221,153,101,155,167, 43,172,9,
   129,22,39,253, 19,98,108,110,79,113,224,232,178,185, 112,104,218,246,97,228,
   251,34,242,193,238,210,144,12,191,179,162,241, 81,51,145,235,249,14,239,107,
   49,192,214, 31,181,199,106,157,184, 84,204,176,115,121,50,45,127, 4,150,254,
   138,236,205,93,222,114,67,29,24,72,243,141,128,195,78,66,215,61,156,180
   };

// This stores the global constants
struct GlobalConstants {

  int noiseMapWidth;
  int noiseMapHeight;
  float* noiseMapData;
  short* permutation;

};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstTerrainGenParams

// Debugging macro
#define gpuErrChk() { gpuAssert(__FILE__, __LINE__); }
inline void gpuAssert(const char *file, int line, bool abort=true)
{
    hipError_t code = hipPeekAtLastError();
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// Constructor for class
TerrainGen::TerrainGen() {

  noiseMap = NULL;
  cudaDeviceNoiseMapData = NULL;
  cudaDevicePermutationTable = NULL;

}

// Destructor for class
TerrainGen::~TerrainGen() {
  
  if (noiseMap) {
    delete noiseMap;
  }

  if (cudaDeviceNoiseMapData) {
    hipFree(cudaDeviceNoiseMapData);
    hipFree(cudaDevicePermutationTable);
  }
}

// ADD MORE METHODS HERE

// kernelClearNoiseMap --  (CUDA device code)
//
// Clear the noise map, setting all pixels to the specified color rgba
__global__ void kernelClearNoiseMap(float h) {

    int noiseMapX = blockIdx.x * blockDim.x + threadIdx.x;
    int noiseMapY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstTerrainGenParams.noiseMapWidth;
    int height = cuConstTerranGenParams.noiseMapHeight;

    if (noiseMapX >= width || noiseMapY >= height)
        return;

    int offset = (noiseMapY * width + noiseMapX);
    float value = h;

    // Write to global memory
    *(float*)(&cuConstTerrainGenParams.noiseMapData[offset]) = value;
}

const noiseMap* TerrainGen::getNoiseMap() {

    // Need to copy contents of the generated noiseMap from device memory
    // before we expose the noiseMap object to the caller

    printf("Copying noise map data from device\n");

    hipMemcpy(noiseMap->data,
               cudaDeviceNoiseMapData,
               sizeof(float) * noiseMap->width * noiseMap->height,
               hipMemcpyDeviceToHost);

    return noiseMap;
}

void TerrainGen::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for TerrainGen\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // Copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy
    hipMalloc(&cudaDeviceNoiseMapData, sizeof(float) * noiseMap->width * noiseMap->height);
    hipMalloc(&cudaDevicePermutationTable, sizeof(short) * 256);

    hipMemcpy(cudaDevicePermutationTable, permutationGlobal,
               sizeof(short) * 256, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.noiseMapWidth = noiseMap->width;
    params.noiseMapHeight = noiseMap->height;
    params.noiseMapData = cudaDeviceNoiseMapData;
    params.permutation = cudaDevicePermutationTable;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstTerrainGenParams), &params, sizeof(GlobalConstants));

}

// allocOutputNoiseMap --
//
// Allocate buffer where we'll put the noise map on the CPU.  Check status of
// noise map first to avoid memory leak.
// Also sets all squares to 0.
void TerrainGen::allocOutputNoiseMap(int width, int height) {

    if (noiseMap)
        delete noiseMap;
    noiseMap = new NoiseMap(width, height);
    noiseMap.clear(0.f); // Set all squares to 0
}

// clearNoiseMapDevice --
//
// Clear the generated noise map, on the device.
void TerrainGen::clearNoiseMapDevice() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (noiseMap->width + blockDim.x - 1) / blockDim.x,
        (noiseMap->height + blockDim.y - 1) / blockDim.y);

    kernelClearNoiseMap<<<gridDim, blockDim>>>(0.f);
    hipDeviceSynchronize();
}

// Interpolation, using the smootherstep() function
// first and second derivatives are both 0
__device__ __inline__ float interpolate(float a0, float a1, float w) {

  return (a1 - a0) * (w * w * w * (w * (w * 6 - 15) + 10)) + a0;
}

// Kernel that generates the Perlin noise map on the GPU
__global__ void perlin(int noiseMapWidth, int noiseMapHeight,
                       int initialGridSize, int octaves, int persistence,
                       int lacunarity, int blockSize) {

  int threadIndex = threadIdx.y * blockDim.y + threadIdx.x;
  int gridSize = initialGridSize;

  __shared__ short permutationTable[256];
  __shared__ float gradients[(blockSize + 1) * (blockSize + 1)];

  if (threadIndex < 256) {
    permutationTable[threadIndex] = permutation[threadIndex];
  }

  for (int i = 0; i < octaves; i++) {

    int gridLeftCoord = (blockSize * blockIdx.x) / gridSize;
    int gridRightCoord = (blockSize * (blockIdx.x + 1)) / gridSize;
    int gridTopCoord = (blockSize * blockIdx.y) / gridSize;
    int gridBottomCoord = (blockSize * (blockIdx.y + 1)) / gridSize;
    int gridNumber = (gridRightCoord - gridLeftCoord) * (gridBottomCoord - gridTopCoord); 

  }

}

// NOTES
/*
  Grid edges will be located at edges of pixels. We will use pixel centers
  to compute the offset vectors.
*/ 

// Main function that generates the terrain. Makes all of the necessary
// kernel calls
void TerrainGen::generate(int initialGridSize, int octaves, int persistence, 
                          int lacunarity) {
  // Call perlin() here, maybe other kernels too

  int noiseMapWidth = cuConstTerrainGenParams.noiseMapWidth;
  int noiseMapHeight = cuConstTerrainGenParams.noiseMapHeight;

  const int blockSize = 100;
  const int threadX = 32;
  const int threadY = 32;
  const int blockX = (noiseMapWidth / 100) + 1;
  const int blockY = (noiseMapHeight / 100) + 1;

  dim3 threadsPerBlock(threadX, threadY, 1);
  dim3 numBlocks(blockX, blockY, 1);
  perlin<< numBlocks, threadsPerBlock >> (noiseMapWidth, noiseMapHeight,
                            initialGridSize, octaves, persistence, lacunarity,
                            blockSize);
}